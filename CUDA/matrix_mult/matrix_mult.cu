
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 100
const int TILE_WIDTH = 16;

__global__
void rand_matrix(float* M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        // curandState_t state;
        // curand_init(0, 0, 0, &state);
        M[row * N + col] = (col + row * col) % MAX;//curand(&state) % MAX;
    }
}

__global__
void mtrx_mult(float* d_M, float* d_N, float* d_P, int Width) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;    int by = blockIdx.y;
    int tx = threadIdx.x;   int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;
    
    for (int ph = 0; ph < Width / TILE_WIDTH; ++ph) {
        Mds[ty][tx] = d_M[Row * Width + ph * TILE_WIDTH + tx];
        Nds[ty][tx] = d_N[(ph * TILE_WIDTH + ty) * Width + Col];
        __syncthreads();

        for(int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();
    }
    d_P[Row * Width + Col] = Pvalue;
}

__global__
void matrixMultiplicationKernel(float* A, float* B, float* C, int N) {
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    C[ROW * N + COL] = tmpSum;
}

__host__
void print_matrix(float* M, int N) {
    printf("============================================\n");
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            printf("%.4f\t", M[i * N + j]);
        }
        printf("\n");
    }
    printf("============================================\n\n");
}

int main(void) {
    int N = 16;  // int N = 1<<20;
    float *A,
          *B,
          *C,
          *d_A,
          *d_B,
          *d_C;
    
    A = (float*)malloc(N * N * sizeof(float));
    B = (float*)malloc(N * N * sizeof(float));
    C = (float*)malloc(N * N * sizeof(float));

    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));
/*******************************************************/
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
    if (N*N > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
    }
/****************FILLING RANDOM MATRIX******************/    
    rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);
    rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_B, N);
    // rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_C, N);
/***************PRINTING RANDOM MATRICES****************/
    hipMemcpy(A, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B, d_B, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // print_matrix(A, N);
    // print_matrix(B, N);
/*******************************************************/
    hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
    hipEventRecord(start,0);

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);

    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    // print_matrix(C, N);
    printf("N: %d ms\n", N);
    printf("SM Runtime : %f ms\n", elapsedTime);
/*******************************************************/
	hipEventCreate(&start);
    hipEventRecord(start,0);

    mtrx_mult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);

    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    // print_matrix(C, N);
    printf("Runtime : %f ms\n", elapsedTime);
/*******************************************************/
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}