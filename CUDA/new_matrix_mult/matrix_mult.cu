#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

using namespace std;

#define MAX 200
#define TILE_WIDTH 32

__global__
void rand_matrix(float* M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        M[row * N + col] = (col + row * col) % MAX;
    }
}

__host__
void print_matrix(float* M, int N) {
    printf("============================================\n");
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            printf("%.4f\t", M[i * N + j]);
        }
        printf("\n");
    }
    printf("============================================\n\n");
}

__global__
void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_NN[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * 2 * TILE_WIDTH + tx;

    float Pvalue = 0; float Pvalue2 = 0;
    // Loop over the M and N tiles required to compute the P element
    if((Row < Width) && (Col < Width)){
        for (int ph = 0; ph < Width / TILE_WIDTH; ++ph){

            // Collaborative loading of M and N tiles into shared memory
            ds_M[ty][tx] = M[Row * Width + ph * TILE_WIDTH + tx];    // ph = tile index
            ds_N[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width + Col];
            ds_NN[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width +(Col + TILE_WIDTH)];
            __syncthreads();

            for (int i = 0; i < TILE_WIDTH; ++i) {
                Pvalue += ds_M[ty][i] * ds_N[i][tx];
                Pvalue2 += ds_M[ty][i] * ds_NN[i][tx];
            }

            __syncthreads();
        }

        P[Row * Width + Col] = Pvalue;
        P[(Row * Width + Col) + TILE_WIDTH] = Pvalue2;
    }
}

void shared_matrix_mult(float *A, float *B, float *C, int N) {
    float *d_A,
          *d_B,
          *d_C;

    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));
/*******************************************************/
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
    if (N * N > 1024) {
        threadsPerBlock.x = 1024;
        threadsPerBlock.y = 1024;
        blocksPerGrid.x = ceil(double(N) / double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(N) / double(threadsPerBlock.y));
    }
/****************FILLING RANDOM MATRIX******************/    
    rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);
    rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_B, N);
/*******************************************************/
    dim3 dimGrid(ceil(N / TILE_WIDTH) / 2, ceil(N / TILE_WIDTH), 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    
    hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
    hipEventRecord(start, 0);

    MatrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
    
    hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    print_matrix(C, N);
    printf("N: %d\n", N);
    printf("SM Runtime : %f ms\n", elapsedTime);
    
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


int main(int argv, char ** argc) {
    long N = 1024;
    float *A,
          *B,
          *C;
    
    A = (float*)malloc(N * N * sizeof(float));
    B = (float*)malloc(N * N * sizeof(float));
    C = (float*)malloc(N * N * sizeof(float));

    for(long long i = 0; i < N * N; i++) {
        A[i] = 3;
        B[i] = 3;
    }

    shared_matrix_mult(A, B, C, N);
    // print_matrix(C, N);
    
    free(A);
    free(B);
    free(C);
}
