
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

using namespace std;

#define TILE_WIDTH 32

__host__
void print_matrix(float* M, int N) {
    printf("============================================\n");
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            printf("%.4f\t", M[i * N + j]);
        }
        printf("\n");
    }
    printf("============================================\n\n");
}

__global__
void MatrixMulKernel(float* M, float* N, float* P,int Width) {
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_NN[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * 2 * TILE_WIDTH + tx;

    float Pvalue = 0; float Pvalue2 = 0;
    // Loop over the M and N tiles required to compute the P element
    if((Row < Width) && (Col < Width)){
      for (int ph = 0; ph < Width/TILE_WIDTH; ++ph){

        // Collaborative loading of M and N tiles into shared memory
        ds_M[ty][tx] = M[Row * Width + ph * TILE_WIDTH + tx];    // ph = tile index
        ds_N[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width + Col];
        ds_NN[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width +(Col + TILE_WIDTH)];
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i) {
            Pvalue += ds_M[ty][i] * ds_N[i][tx];
            Pvalue2 += ds_M[ty][i] * ds_NN[i][tx];
        }

        __syncthreads();
      }

      P[Row * Width + Col] = Pvalue;
      P[(Row * Width + Col) + TILE_WIDTH] = Pvalue2;
    }
}

void print(float *M, int n) {
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < n; j++) {
            cout<<M[n * i + j]<<" ";
        }
        cout<<endl;
    }
    cout<<endl;
}

void Mul_tiled(float *A, float *B, float *C, int n) {
    long long size = sizeof(float) *n*n;
    float *d_A,
          *d_B,
          *d_C;

    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_B, size);
    hipMalloc((void **) &d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(n / TILE_WIDTH) / 2, ceil(n / TILE_WIDTH), 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    // Prepare
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop); 
    // Start record
    hipEventRecord(start, 0);
    MatrixMulKernel<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, n);
    // Stop event
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    // Clean up:
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    printf("Runtime : %f ms\n", elapsedTime);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


int main(int argv, char ** argc) {
    long N = 1024;
    float *A,
          *B,
          *C;
    
    A = (float*)malloc(N * N * sizeof(float));
    B = (float*)malloc(N * N * sizeof(float));
    C = (float*)malloc(N * N * sizeof(float));

    for(long long i = 0; i < N * N; i++) {
        A[i] = 3;
        B[i] = 3;
    }

    Mul_tiled(A, B, C, N);
    // print_matrix(C, N);
    
    free(A);
    free(B);
    free(C);
}
