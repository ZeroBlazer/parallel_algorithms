
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

using namespace std;

#define MAX 200;
#define TILE_WIDTH 32

__global__
void rand_matrix(float* M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        M[row * N + col] = (col + row * col) % MAX;
    }
}

__host__
void print_matrix(float* M, int N) {
    printf("============================================\n");
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            printf("%.4f\t", M[i * N + j]);
        }
        printf("\n");
    }
    printf("============================================\n\n");
}

__global__
void MatrixMulKernel(float* M, float* N, float* P,int Width) {
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_NN[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * 2 * TILE_WIDTH + tx;

    float Pvalue = 0; float Pvalue2 = 0;
    // Loop over the M and N tiles required to compute the P element
    if((Row < Width) && (Col < Width)){
      for (int ph = 0; ph < Width/TILE_WIDTH; ++ph){

        // Collaborative loading of M and N tiles into shared memory
        ds_M[ty][tx] = M[Row * Width + ph * TILE_WIDTH + tx];    // ph = tile index
        ds_N[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width + Col];
        ds_NN[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width +(Col + TILE_WIDTH)];
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i) {
            Pvalue += ds_M[ty][i] * ds_N[i][tx];
            Pvalue2 += ds_M[ty][i] * ds_NN[i][tx];
        }

        __syncthreads();
      }

      P[Row * Width + Col] = Pvalue;
      P[(Row * Width + Col) + TILE_WIDTH] = Pvalue2;
    }
}

void Mul_tiled(float *A, float *B, float *C, int N) {
    long long size = sizeof(float) * N * N;
    float *d_A,
          *d_B,
          *d_C;

    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_B, size);
    hipMalloc((void **) &d_C, size);
/*******************************************************/
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
    if (N*N > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
    }
/****************FILLING RANDOM MATRIX******************/    
    rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);
    rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_B, N);
/*******************************************************/
    dim3 dimGrid(ceil(N / TILE_WIDTH) / 2, ceil(N / TILE_WIDTH), 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    
    hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
    hipEventRecord(start, 0);

    MatrixMulKernel<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
    
    hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);

    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    // print_matrix(C, N);
    printf("N: %d\n", N);
    printf("SM Runtime : %f ms\n", elapsedTime);
    
    // Clean up:
    // cudaEventDestroy(start);
    // cudaEventDestroy(stop);
    
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


int main(int argv, char ** argc) {
    long N = 1024;
    float *A,
          *B,
          *C;
    
    A = (float*)malloc(N * N * sizeof(float));
    B = (float*)malloc(N * N * sizeof(float));
    C = (float*)malloc(N * N * sizeof(float));

    for(long long i = 0; i < N * N; i++) {
        A[i] = 3;
        B[i] = 3;
    }

    Mul_tiled(A, B, C, N);
    // print_matrix(C, N);
    
    free(A);
    free(B);
    free(C);
}
