
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_THREADS 16

__global__
void grayScale(int * img, int * res, int altura, int ancho){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(col < ancho && row < altura){
		int r = img[altura * ancho * 0 + row * ancho + col];
		int g = img[altura * ancho * 1 + row * ancho + col];
		int b = img[altura * ancho * 2 + row * ancho + col];
		int gray = 0.21 * r + 0.72 * g + 0.07 * b;
		res[row * ancho + col] = gray;
	}
}

void hostGrayScale(int * img, int * res, int altura, int ancho){
	int size = sizeof(int) * altura * ancho;
	int * d_img;
	int * d_res;
	hipMalloc((void**) &d_img, size * 3);
	hipMalloc((void**) &d_res, size);
	hipMemcpy(d_img, img, size * 3, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(ancho/NUM_THREADS),ceil(altura/NUM_THREADS),1);
	dim3 dimBlock(NUM_THREADS,NUM_THREADS,1);

	grayScale<<<dimGrid,dimBlock>>>(d_img,d_res,altura,ancho);

	hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);
	hipFree(d_img);
	hipFree(d_res);
}

int * loadImage(char * file, int * altura, int * ancho){
	FILE * archivo = fopen(file,"r");
	char * line;
	size_t len = 0;
	getline(&line,&len,archivo);
	*altura = atoi(line);
	getline(&line,&len,archivo);
	*ancho = atoi(line);
	int * res = (int *) malloc(sizeof(int) * (*ancho) * (*altura) * 3);
	for(int i = 0; i < 3; i++){
		for(int j = 0; j < *altura; j++){
			for(int k = 0; k < *ancho; k++){
				getline(&line,&len,archivo);
				res[(*altura) * (*ancho) * i + j * (*ancho) + k] = atoi(line);
			}
		}
	}
	fclose(archivo);
	return res;
}

void saveImage(int * res, char * file, int altura, int ancho){
	FILE * archivo = fopen(file,"w");
	fprintf(archivo, "%d\n", altura);
	fprintf(archivo, "%d\n", ancho);
	for(int i = 0; i < altura; i++){
		for(int j = 0; j < ancho; j++){
			fprintf(archivo, "%d\n", res[i * ancho + j]);
		}
	}
	fclose(archivo);
}

void print(int * res, int altura, int ancho){
	for(int i = 0; i < 3; i++){
		for(int j = 0; j < altura; j++){
			for(int k = 0; k < ancho; k++){
				printf("%d\n", res[altura * ancho * i + j * ancho + k]);
			}
		}
	}
}


int main(int argv, char ** argc){
	if(argv != 2){
		printf("Faltan Argumentos <img>\n");
		return 0;
	}
	char * fileImg = argc[1];
    char * programImg = "./img ";
	char * temp = " 0 >> temp"; 
	char * comp = (char *) malloc(strlen(programImg) + strlen(fileImg) + strlen(temp));
	comp = strcpy(comp, programImg);
	comp = strcat(comp, fileImg);
	comp = strcat(comp, temp);
	printf("%s\n", comp);
	system(comp);
	int altura = 0;
	int ancho = 0;
	int * img = loadImage("temp",&altura, &ancho);
	system("rm temp");
	saveImage(img,"res",altura,ancho);
	system("./img res 1");
	system("rm res");
}
