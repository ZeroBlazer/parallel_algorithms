
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 100

__global__
void rand_matrix(float* M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        hiprandState_t state;
        hiprand_init(0, 0, 0, &state);
        M[row * N + col] = hiprand(&state) % MAX;
    }
}

// __global__
// void mtrx_sum_elem(float* C, float* A, float* B, int N) {
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     int col = blockIdx.x * blockDim.x + threadIdx.x;

// }

// __global__
// void mtrx_sum_row(float* C, float* A, float* B, int N) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;

// }

// __global__
// void mtrx_sum_column(float* C, float* A, float* B, int N) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;

// }

__host__
void print_matrix(float* M, int N) {
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            printf("%d\t", M[i * N + j]);
        }
        printf("\n");
    }
}

int main(void) {
    int N = 5;  // int N = 1<<20;
    float *A,
          *B,
          *C,
          *d_A,
          *d_B,
          *d_C;
    
    A = (float*)malloc(N * N * sizeof(float));
    B = (float*)malloc(N * N * sizeof(float));
    C = (float*)malloc(N * N * sizeof(float));

    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));
/****************FILLING RANDOM MATRIX******************/    
    rand_matrix<<<ceil(N/256.0), 256>>>(d_A, N);
    rand_matrix<<<ceil(N/256.0), 256>>>(d_B, N);
    rand_matrix<<<ceil(N/256.0), 256>>>(d_C, N);
/***************PRINTING RANDOM MATRIX******************/
    hipMemcpy(A, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B, d_B, N * N * sizeof(float), hipMemcpyDeviceToHost);

    print_matrix(A, N);
    print_matrix(B, N);
/*******************************************************/
    
//   cudaMemcpy(d_x, x, N*sizeof(float), cudaMemcpyHostToDevice);
//   cudaMemcpy(d_y, y, N*sizeof(float), cudaMemcpyHostToDevice);

//   // Perform SAXPY on 1M elements
//   saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

//   cudaMemcpy(y, d_y, N*sizeof(float), cudaMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(A);
  free(B);
  free(C);
}