
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 100

__global__
void rand_matrix(float* M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        hiprandState_t state;
        hiprand_init(0, 0, 0, &state);
        M[row * N + col] = hiprand(&state) % MAX;
    }
}

__global__
void mtrx_sum_elem(float* C, float* A, float* B, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < N && col < N) {
        C[row * N + col] = A[row * N + col] + B[row * N + col];
    }
}

__global__
void mtrx_sum_row(float* C, float* A, float* B, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < N) {
        for(int i = 0; i < N; ++i) {
            C[id * N + i] = A[id * N + i] + B[id * N + i];
        }
    }
}

// __global__
// void mtrx_sum_column(float* C, float* A, float* B, int N) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;

// }

__host__
void print_matrix(float* M, int N) {
    printf("============================================\n");
    for(int i = 0; i < N; ++i) {
        for(int j = 0; j < N; ++j) {
            printf("%.4f\t", M[i * N + j]);
        }
        printf("\n");
    }
    printf("============================================\n\n");
}

int main(void) {
    int N = 5;  // int N = 1<<20;
    float *A,
          *B,
          *C,
          *d_A,
          *d_B,
          *d_C;
    
    A = (float*)malloc(N * N * sizeof(float));
    B = (float*)malloc(N * N * sizeof(float));
    C = (float*)malloc(N * N * sizeof(float));

    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));
/*******************************************************/
    dim3 threadsPerBlock(N, N);
    dim3 blocksPerGrid(1, 1);
    if (N*N > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
    }
/****************FILLING RANDOM MATRIX******************/    
    rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);
    rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_B, N);
    rand_matrix<<<blocksPerGrid, threadsPerBlock>>>(d_C, N);
/***************PRINTING RANDOM MATRICES****************/
    hipMemcpy(A, d_A, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B, d_B, N * N * sizeof(float), hipMemcpyDeviceToHost);

    print_matrix(A, N);
    print_matrix(B, N);
/*******************************************************/
    mtrx_sum_elem<<<blocksPerGrid, threadsPerBlock>>>(d_C, d_A, d_C, N);
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    print_matrix(C, N);
/*******************************************************/
    mtrx_sum_row<<<ceil(N * N / 512.0), 512>>>(d_C, d_A, d_C, N);
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    print_matrix(C, N);
/*******************************************************/
//     mtrx_sum_column<<<ceil(N/256.0), 256>>>(d_C, d_A, d_C, N);
//     cudaMemcpy(C, d_C, N * N * sizeof(float), cudaMemcpyDeviceToHost);

//     print_matrix(C, N);
/*******************************************************/
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(A);
    free(B);
    free(C);
}